/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

#include <hip/hip_runtime.h>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& means3D,
    const torch::Tensor& intensity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const float viewdepth,
	const int image_channel,
    const int image_height,
    const int image_width,
	const bool prefiltered,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }

  hipSetDevice(means3D.device().index());
  
  const int P = means3D.size(0);
  const int C = image_channel;
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_intensity = torch::full({C, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P,
		C, W, H,
		viewdepth,
		means3D.contiguous().data<float>(),
		intensity.contiguous().data<float>(),  
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		prefiltered,
		out_intensity.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_intensity, radii, geomBuffer, binningBuffer, imgBuffer);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& intensity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& dL_dout_intensity,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool debug) 
{
  hipSetDevice(means3D.device().index());

  const int P = means3D.size(0);
  const int C = dL_dout_intensity.size(0);
  const int H = dL_dout_intensity.size(1);
  const int W = dL_dout_intensity.size(2);

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dintensity= torch::zeros({P, C}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, R,
	  C, W, H, 
	  intensity.contiguous().data<float>(),
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_intensity.contiguous().data<float>(),
	  dL_dmeans2D.contiguous().data<float>(),
	  dL_dconic.contiguous().data<float>(),  
	  dL_dintensity.contiguous().data<float>(),
	  dL_dmeans3D.contiguous().data<float>(),
	  dL_dcov3D.contiguous().data<float>(),
	  dL_dscales.contiguous().data<float>(),
	  dL_drotations.contiguous().data<float>(),
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dintensity, dL_dmeans3D, dL_dcov3D, dL_dscales, dL_drotations);
}

std::tuple<torch::Tensor, torch::Tensor>
// torch::Tensor
 markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		const int H, const int W,
		const float viewdepth)
{ 
  hipSetDevice(means3D.device().index());
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
  torch::Tensor proj_points = torch::zeros({P, 3}, means3D.options());
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		H, W, 
		viewdepth,
		present.contiguous().data<bool>(),
		proj_points.contiguous().data<float>());
  }
  
  return std::make_tuple(present, proj_points);
//   return present;
}